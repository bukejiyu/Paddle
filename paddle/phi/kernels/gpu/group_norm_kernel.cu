#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/group_norm_kernel.h"

#include "paddle/common/layout.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpu/group_norm_utils.h"

#include "paddle/phi/common/data_type.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/device_context.h"

namespace phi {

static inline int32_t divUp(int32_t m, int32_t n) { return (m + n - 1) / n; }

static inline __device__ __host__ float sigmoid(float x) {
  return 1.F / (1.F + expf(-x));
}

#ifdef PADDLE_CUDA_BF16
__host__ __device__ inline float2 bfloat1622float2(const __hip_bfloat162 a) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
  return __bfloat1622float2(a);
#else
  float hi_float;
  float lo_float;
  lo_float = __internal_bfloat162float(((__hip_bfloat162_raw)a).x);
  hi_float = __internal_bfloat162float(((__hip_bfloat162_raw)a).y);
  return make_float2(lo_float, hi_float);
#endif
}

__host__ __device__ inline __hip_bfloat162 float22bfloat162_rn(const float2 a) {
  __hip_bfloat162 val;
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
  val = __float22bfloat162_rn(a);
#else
  val.x = __float2bfloat16_rn(a.x);
  val.y = __float2bfloat16_rn(a.y);
#endif
  return val;
}

#endif

template <typename T>
__host__ __device__ inline float __2float(const T a) {
  return static_cast<float>(a);
}

template <>
__host__ __device__ inline float __2float<__half>(const __half a) {
  return __half2float(a);
}

template <typename T>
__host__ __device__ inline T __2dst(const float a) {
  return static_cast<T>(a);
}

template <>
__host__ __device__ inline __half __2dst<__half>(const float a) {
  return __float2half(a);
}

struct GroupSums {
  // Is it the 1st element of the group?
  int32_t flag;
  // The sum.
  float sum;
  // The sum of squares.
  float sumSq;
};

struct GroupSumsOp {
  inline __device__ GroupSums operator()(GroupSums const& a,
                                         GroupSums const& b) {
    GroupSums dst;
    dst.sum = b.flag ? b.sum : (a.sum + b.sum);
    dst.sumSq = b.flag ? b.sumSq : (a.sumSq + b.sumSq);
    dst.flag = a.flag + b.flag;
    return dst;
  }
};

static int32_t findMaxDivisor(int32_t n, int32_t maxAllowedDivisor) {
  int32_t maxDivisor = -1;
  for (int32_t i = 1; i <= std::sqrt(n); i++) {
    if (n % i == 0) {
      int32_t divisor1 = n / i;
      int32_t divisor2 = i;

      if (divisor1 > maxDivisor && divisor1 < maxAllowedDivisor) {
        maxDivisor = divisor1;
      }
      if (divisor2 > maxDivisor && divisor2 < maxAllowedDivisor) {
        maxDivisor = divisor2;
      }
    }
  }
  return maxDivisor;
}

template <typename T, int THREADS_PER_CHANNEL>
inline __device__ void UpdateSum(const T* srcX, float* sum, float* sumSq) {
  float src_data = phi::__2float<T>(*srcX);
  *sum += src_data;
  *sumSq += src_data * src_data;
}

template <typename T, int THREADS_PER_CHANNEL>
inline __device__ void UpdateSum(const T* srcX,
                                 const T* srcY,
                                 float* sum,
                                 float* sumSq) {
  float src_data = phi::__2float<T>(*srcX);
  float srcy_data = phi::__2float<T>(*srcY);
  *sum += src_data + srcy_data;
  *sumSq += (src_data + srcy_data) * (src_data + srcy_data);
}

template <>
inline __device__ void UpdateSum<__half, 2>(const __half* srcX,
                                            float* sum,
                                            float* sumSq) {
  __half2 h2 = *reinterpret_cast<__half2 const*>(srcX);
  float2 f2 = __half22float2(h2);
  *sum += f2.x + f2.y;
  *sumSq += f2.x * f2.x + f2.y * f2.y;
}

template <>
inline __device__ void UpdateSum<__half, 2>(const __half* srcX,
                                            const __half* srcY,
                                            float* sum,
                                            float* sumSq) {
  __half2 h2 = *reinterpret_cast<__half2 const*>(srcX);
  __half2 h2_r = *reinterpret_cast<__half2 const*>(srcY);
  float2 f2 = __half22float2(h2);
  float2 f2_r = __half22float2(h2_r);
  *sum += f2.x + f2_r.x + f2.y + f2_r.y;
  ;
  *sumSq +=
      (f2.x + f2_r.x) * (f2.x + f2_r.x) + (f2.y + f2_r.y) * (f2.y + f2_r.y);
}

template <>
inline __device__ void UpdateSum<phi::dtype::float16, 2>(
    const phi::dtype::float16* srcX, float* sum, float* sumSq) {
  __half2 h2 = *reinterpret_cast<__half2 const*>(srcX);
  float2 f2 = __half22float2(h2);
  *sum += f2.x + f2.y;
  *sumSq += f2.x * f2.x + f2.y * f2.y;
}

template <>
inline __device__ void UpdateSum<phi::dtype::float16, 2>(
    const phi::dtype::float16* srcX,
    const phi::dtype::float16* srcY,
    float* sum,
    float* sumSq) {
  __half2 h2 = *reinterpret_cast<__half2 const*>(srcX);
  __half2 h2_r = *reinterpret_cast<__half2 const*>(srcY);
  float2 f2 = __half22float2(h2);
  float2 f2_r = __half22float2(h2_r);
  *sum += f2.x + f2_r.x + f2.y + f2_r.y;
  *sumSq +=
      (f2.x + f2_r.x) * (f2.x + f2_r.x) + (f2.y + f2_r.y) * (f2.y + f2_r.y);
}

#ifdef PADDLE_CUDA_BF16
template <>
inline __device__ void UpdateSum<phi::dtype::bfloat16, 2>(
    const phi::dtype::bfloat16* srcX, float* sum, float* sumSq) {
  __hip_bfloat162 h2 = *reinterpret_cast<__hip_bfloat162 const*>(srcX);
  float2 f2 = phi::bfloat1622float2(h2);
  *sum += f2.x + f2.y;
  *sumSq += f2.x * f2.x + f2.y * f2.y;
}

template <>
inline __device__ void UpdateSum<phi::dtype::bfloat16, 2>(
    const phi::dtype::bfloat16* srcX,
    const phi::dtype::bfloat16* srcY,
    float* sum,
    float* sumSq) {
  __hip_bfloat162 h2 = *reinterpret_cast<__hip_bfloat162 const*>(srcX);
  __hip_bfloat162 h2_r = *reinterpret_cast<__hip_bfloat162 const*>(srcY);
  float2 f2 = phi::bfloat1622float2(h2);
  float2 f2_r = phi::bfloat1622float2(h2_r);
  *sum += f2.x + f2_r.x + f2.y + f2_r.y;
  *sumSq +=
      (f2.x + f2_r.x) * (f2.x + f2_r.x) + (f2.y + f2_r.y) * (f2.y + f2_r.y);
}
#endif

// grid是一样的 x:是gid,或者是 blockid，y是dhw的 最大除数，z是n
// 线程 按照 cperblock 来
template <typename T, int THREADS_PER_BLOCK>
__global__ void groupNormNDHWCSumSingerChannelKernel(
    const GroupNormNDHWCParams<T> params) {
  // The instance in the batch.
  __shared__ float2 smem[THREADS_PER_BLOCK];
  int32_t ni = blockIdx.z;
  int32_t ci = blockIdx.x * params.cPerBlock + threadIdx.x;
  if (ci >= params.c) {
    return;
  }
  // The first activation loaded by that block.
  int32_t dhwBegin = blockIdx.y * params.dhwPerBlock;
  // The last activation loaded by that block.
  int32_t dhwEnd = min(dhwBegin + params.dhwPerBlock, params.dhw);

  // The sums.
  float sum = 0.F;
  float sumSq = 0.F;

  for (int32_t dhwi = dhwBegin; dhwi < dhwEnd; ++dhwi) {
    // The offset.
    int64_t offset = static_cast<int64_t>(ni) * params.dhwc +
                     static_cast<int64_t>(dhwi) * params.c + ci;
    float src_data = *reinterpret_cast<float const*>(&params.srcX[offset]);
    if (params.srcY != nullptr) {
      UpdateSum<T, 1>(&params.srcX[offset], &params.srcY[ci], &sum, &sumSq);
    } else {
      UpdateSum<T, 1>(&params.srcX[offset], &sum, &sumSq);
    }
  }

  smem[threadIdx.x] = make_float2(sum, sumSq);

  __syncthreads();

  float2 sums = smem[threadIdx.x];
  atomicAdd(&params.redBuffer[(2 * ni + 0) * params.groups + ci],
            sums.x * params.invDHWC);
  atomicAdd(&params.redBuffer[(2 * ni + 1) * params.groups + ci], sums.y);
}

// THREADS_PER_BLOCK ：每个block的线程数
// THREADS_PER_CHANNEL:每个线程处理的 groupsize个数
// blocksPerDHW 不大于1024 且是 dhw的最大除数
// 大概率是groups 其实是需要用多少block处理c ，
// grid(groups,是dhw 的最大除数，不能大于1024 y=blocksPerDHW:每个block
// 需要处理的 dhw 数,n) thread(THREADS_PER_BLOCK,1,1) cPerBlock grid
// 需要多少个block.x处理 c 需要用多少个block 处理 dhw 需要n个block 处理 n
template <typename T, int THREADS_PER_BLOCK, int THREADS_PER_CHANNEL>
__global__ void groupNormNDHWCSumKernel(const GroupNormNDHWCParams<T> params) {
  // The object in charge of doing the sums for the different blocks.
  typedef hipcub::BlockScan<GroupSums, THREADS_PER_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage tempStorage;
  // Allocate shared memory for BlockScan.
  // Allocate shared memory for the groups. We could reduce the amount of shared
  // memory reserved.
  __shared__ float2 smem[THREADS_PER_BLOCK];

  // The instance in the batch.
  int32_t ni = blockIdx.z;
  // The channel loaded by that thread (2 channels per thread for F16x2).
  int32_t ci =
      blockIdx.x * params.cPerBlock + threadIdx.x * THREADS_PER_CHANNEL;
  if (ci >= params.c || threadIdx.x * THREADS_PER_CHANNEL >= params.cPerBlock) {
    return;
  }
  // gj 是全局ci 属于那个 gid
  int32_t gj = ci / params.cPerGroup;
  // cj 是全局ci 属于那个 gsize_id
  int32_t cj = ci % params.cPerGroup;
  // The first activation loaded by that block.
  int32_t dhwBegin = blockIdx.y * params.dhwPerBlock;
  // The last activation loaded by that block.
  // 不能超过我要处理的 dhw
  int32_t dhwEnd = min(dhwBegin + params.dhwPerBlock, params.dhw);

  // The sums.
  float sum = 0.F;
  float sumSq = 0.F;

  for (int32_t dhwi = dhwBegin; dhwi < dhwEnd; ++dhwi) {
    // The offset.
    int64_t offset = static_cast<int64_t>(ni) * params.dhwc +
                     static_cast<int64_t>(dhwi) * params.c + ci;
    float src_data = *reinterpret_cast<float const*>(&params.srcX[offset]);
    if (params.srcY != nullptr) {
      UpdateSum<T, THREADS_PER_CHANNEL>(
          &params.srcX[offset],
          &params.srcY[gj * params.cPerGroup + cj],
          &sum,
          &sumSq);
    } else {
      UpdateSum<T, THREADS_PER_CHANNEL>(&params.srcX[offset], &sum, &sumSq);
    }
  }

  // The group that thread works on and the channel in the group (modulus).
  // gi是 每个groupsize 在 blocksize中的偏移，他是block中的第几个 groupid
  // [g0][g1]
  // [  b0  ] 一个block中如果处理了2个g  gi是在定位是block中的第几个g
  int32_t gi =
      ci / params.cPerGroup - blockIdx.x * params.cPerBlock / params.cPerGroup;
  // 如果在处理第一个线程 或者 cj是group中的第一个时 设置成 flag=1
  int flag = (cj == 0 || threadIdx.x == 0) ? 1 : 0;
  GroupSums inp{flag, sum, sumSq};
  GroupSums out;
  BlockScan(tempStorage).InclusiveScan(inp, out, GroupSumsOp());
  // 同时进行扫描，结果会存在 out中，因为时前缀合，并且为了应对
  // 一个block中处理了多个group的情况，每个group开头都会//这个scan是会
  // 扫描停止的么？ 这是同步的么？是按顺序扫描的吧？

  // 最后一个groupsize 和 或者最后block中处理的最后一个线程 会存储起来
  if (cj == params.cPerGroup - THREADS_PER_CHANNEL ||
      threadIdx.x * THREADS_PER_CHANNEL ==
          params.cPerBlock - THREADS_PER_CHANNEL) {
    smem[gi] = make_float2(out.sum, out.sumSq);
  }

  __syncthreads();

  if (cj == params.cPerGroup - THREADS_PER_CHANNEL ||
      threadIdx.x * THREADS_PER_CHANNEL ==
          params.cPerBlock - THREADS_PER_CHANNEL) {
    float2 sums = smem[gi];
    // 他的存储方式是  g0...gn
    atomicAdd(&params.redBuffer[(2 * ni + 0) * params.groups + gj],
              sums.x * params.invDHWC);
    atomicAdd(&params.redBuffer[(2 * ni + 1) * params.groups + gj], sums.y);
  }
}

template <typename T>
void groupNormNDHWCSum<T>::operator()(GroupNormNDHWCParams<T>* params,
                                      gpuStream_t stream) {
  dim3 grid;
  // cPerBlock 每一个block中有处理多少个 groupsize
  //  如果cPerBlock 能被 group_size 整除就直接使用 ，否则 换成 group_size
  // cPerBlock 每一个block中有处理多少个 groupsize
  // x=c/groupsize（基本上是） ，如果c不能被cperblock整除，会向上取整
  // 大概率这个x是gid cPerBlock c分割的大小
  // 几乎等于groupsize的大小或者groupsize的倍数 所以 x的是c/ 分割的大小
  // 需要多少个block处理 c
  grid.x = divUp(params->c, params->cPerBlock);
  // 不大于 1024的 dhw 能整除的数，最大是1024,  blocksPerDHW
  // 感觉是dhw需要用多少个block处理,但是这个blockperdhw 是dhw的除数呀
  // dhwPerBlock:每个block中有需要处理多少个dhw dhw/blocksPerDHW 一定可以被整除
  // y=blocksPerDHW:每个block 需要处理的 dhw 数
  grid.y = divUp(params->dhw, params->dhwPerBlock);
  grid.z = params->n;
  // cPerBlock: 要么是groupsize的倍数 要么是groupsize
  // cPerGroup: c/groups 如果groupsize 可以被2整除 ，那么用float2去处理 vector
  // 线程直接按 groupsize 取 2或者 取1
  // 为什么 每个block中只处理  groupsize个 或者 groupsize个的倍数？
  if (params->cPerGroup % 2 == 0) {
    switch (params->cPerBlock) {
      case 512:
      case 480:
        groupNormNDHWCSumKernel<T, 256, 2><<<grid, 256, 0, stream>>>(*params);
        break;
      case 320:
        groupNormNDHWCSumKernel<T, 160, 2><<<grid, 160, 0, stream>>>(*params);
        break;
      case 256:
        groupNormNDHWCSumKernel<T, 128, 2><<<grid, 128, 0, stream>>>(*params);
        break;
      case 128:
        groupNormNDHWCSumKernel<T, 64, 2><<<grid, 64, 0, stream>>>(*params);
        break;
      default:
        // 如果c是特殊的 大小 需要向上取整 需要处理remain
        grid.x = divUp(params->c, 128);
        params->cPerBlock = 128;
        groupNormNDHWCSumKernel<T, 64, 2><<<grid, 64, 0, stream>>>(*params);
    }
  } else {
    if (params->cPerGroup != 1) {
      // groupsize 是单数
      switch (params->cPerBlock) {
        case 512:
          groupNormNDHWCSumKernel<T, 512, 1><<<grid, 512, 0, stream>>>(*params);
          break;
        case 480:
          groupNormNDHWCSumKernel<T, 480, 1><<<grid, 480, 0, stream>>>(*params);
          break;
        case 320:
          groupNormNDHWCSumKernel<T, 320, 1><<<grid, 320, 0, stream>>>(*params);
          break;
        case 256:
          groupNormNDHWCSumKernel<T, 256, 1><<<grid, 256, 0, stream>>>(*params);
          break;
        case 128:
          groupNormNDHWCSumKernel<T, 128, 1><<<grid, 128, 0, stream>>>(*params);
          break;
        default:
          grid.x = divUp(params->c, 128);
          params->cPerBlock = 128;
          groupNormNDHWCSumKernel<T, 128, 1><<<grid, 128, 0, stream>>>(*params);
      }
    } else {
      // 这是在说 groupsize=1的情况 退化成 layernorm了感觉
      switch (params->cPerBlock) {
        case 512:
          groupNormNDHWCSumSingerChannelKernel<T, 512>
              <<<grid, 512, 0, stream>>>(*params);
          break;
        case 480:
          groupNormNDHWCSumSingerChannelKernel<T, 480>
              <<<grid, 480, 0, stream>>>(*params);
          break;
        case 320:
          groupNormNDHWCSumSingerChannelKernel<T, 320>
              <<<grid, 320, 0, stream>>>(*params);
          break;
        case 256:
          groupNormNDHWCSumSingerChannelKernel<T, 256>
              <<<grid, 256, 0, stream>>>(*params);
          break;
        case 128:
          groupNormNDHWCSumSingerChannelKernel<T, 128>
              <<<grid, 128, 0, stream>>>(*params);
          break;
        default:
          grid.x = divUp(params->c, 128);
          params->cPerBlock = 128;
          groupNormNDHWCSumSingerChannelKernel<T, 128>
              <<<grid, 128, 0, stream>>>(*params);
      }
    }
  }
}
template class groupNormNDHWCSum<half>;

template <typename T, int THREADS_PER_CHANNEL>
inline __device__ void GroupNormCompute(int32_t dhwBegin,
                                        int32_t dhwEnd,
                                        int32_t ci,  // 在一个block中的id
                                        const GroupNormNDHWCParams<T>& params,
                                        float mean,
                                        float invStdDev) {
  float gamma =
      phi::__2float<T>(*(reinterpret_cast<T const*>(params.gamma) + ci));
  float beta =
      phi::__2float<T>(*(reinterpret_cast<T const*>(params.beta) + ci));
  for (int32_t dhwi = dhwBegin; dhwi < dhwEnd; ++dhwi) {
    // The src/dst offset.
    int64_t offset = (int64_t)blockIdx.z * params.dhwc + dhwi * params.c + ci;
    float src_data = phi::__2float<T>(params.srcX[offset]);
    // 获取残差，如果有的话
    if (params.srcY != nullptr) {
      auto gi = ci / params.cPerGroup;
      auto gj = ci % params.cPerGroup;
      src_data += phi::__2float<T>(params.srcY[gi * params.cPerGroup + gj]);
      *reinterpret_cast<T*>(&params.eleOut[offset]) = phi::__2dst<T>(src_data);
    }
    // Normalize the channels.
    float dst_data = (src_data - mean) * invStdDev;
    // Scale by gamma and add beta.
    dst_data = gamma * dst_data + beta;

    // Apply Silu if needed.
    if (params.withSilu) {
      dst_data = dst_data * sigmoid(dst_data);
    }

    // Store the scaled values.
    *reinterpret_cast<T*>(&params.dst[offset]) = phi::__2dst<T>(dst_data);
  }
}

template <>
inline __device__ void GroupNormCompute<phi::dtype::float16, 2>(
    int32_t dhwBegin,
    int32_t dhwEnd,
    int32_t ci,
    const GroupNormNDHWCParams<phi::dtype::float16>& params,
    float mean,
    float invStdDev) {
  float2 gammaF2, betaF2;
  gammaF2 = __half22float2(*reinterpret_cast<__half2 const*>(
      reinterpret_cast<half const*>(params.gamma) + ci));
  betaF2 = __half22float2(*reinterpret_cast<__half2 const*>(
      reinterpret_cast<half const*>(params.beta) + ci));

  // Iterate over the activations to compute the sums.
  for (int32_t dhwi = dhwBegin; dhwi < dhwEnd; ++dhwi) {
    // The src/dst offset.
    int64_t offset = (int64_t)blockIdx.z * params.dhwc + dhwi * params.c + ci;

    // Fetch two channels per thread.
    __half2 h2 = *reinterpret_cast<__half2 const*>(&params.srcX[offset]);

    // Extract the two half values.
    float2 f2 = __half22float2(h2);

    // 获取残差，如果有的话
    if (params.srcY != nullptr) {
      auto gi = ci / params.cPerGroup;
      auto gj = ci % params.cPerGroup;
      __half2 r2 = *reinterpret_cast<__half2 const*>(
          &params.srcY[gi * params.cPerGroup + gj]);
      float2 r_f2 = __half22float2(r2);
      f2.x += r_f2.x;
      f2.y += r_f2.y;
      *reinterpret_cast<__half2*>(&params.eleOut[offset]) =
          __float22half2_rn(f2);
    }
    // Normalize the channels.
    f2.x = (f2.x - mean) * invStdDev;
    f2.y = (f2.y - mean) * invStdDev;

    // Scale by gamma and add beta.
    f2.x = gammaF2.x * f2.x + betaF2.x;
    f2.y = gammaF2.y * f2.y + betaF2.y;

    // Apply Silu if needed.
    if (params.withSilu) {
      f2.x = f2.x * sigmoid(f2.x);
      f2.y = f2.y * sigmoid(f2.y);
    }
    // Store the scaled values.
    *reinterpret_cast<__half2*>(&params.dst[offset]) = __float22half2_rn(f2);
  }
}

template <>
inline __device__ void GroupNormCompute<__half, 2>(
    int32_t dhwBegin,
    int32_t dhwEnd,
    int32_t ci,
    const GroupNormNDHWCParams<__half>& params,
    float mean,
    float invStdDev) {
  float2 gammaF2, betaF2;
  gammaF2 = __half22float2(*reinterpret_cast<__half2 const*>(
      reinterpret_cast<half const*>(params.gamma) + ci));
  betaF2 = __half22float2(*reinterpret_cast<__half2 const*>(
      reinterpret_cast<half const*>(params.beta) + ci));

  // Iterate over the activations to compute the sums.
  for (int32_t dhwi = dhwBegin; dhwi < dhwEnd; ++dhwi) {
    // The src/dst offset.
    int64_t offset = (int64_t)blockIdx.z * params.dhwc + dhwi * params.c + ci;

    // Fetch two channels per thread.
    __half2 h2 = *reinterpret_cast<__half2 const*>(&params.srcX[offset]);

    // Extract the two half values.
    float2 f2 = __half22float2(h2);
    // 获取残差，如果有的话
    if (params.srcY != nullptr) {
      auto gi = ci / params.cPerGroup;
      auto gj = ci % params.cPerGroup;
      __half2 r2 = *reinterpret_cast<__half2 const*>(
          &params.srcY[gi * params.cPerGroup + gj]);
      float2 r_f2 = __half22float2(r2);
      f2.x += r_f2.x;
      f2.y += r_f2.y;
      *reinterpret_cast<__half2*>(&params.eleOut[offset]) =
          __float22half2_rn(f2);
    }
    // Normalize the channels.
    f2.x = (f2.x - mean) * invStdDev;
    f2.y = (f2.y - mean) * invStdDev;

    // Scale by gamma and add beta.
    f2.x = gammaF2.x * f2.x + betaF2.x;
    f2.y = gammaF2.y * f2.y + betaF2.y;

    // Apply Silu if needed.
    if (params.withSilu) {
      f2.x = f2.x * sigmoid(f2.x);
      f2.y = f2.y * sigmoid(f2.y);
    }
    // Store the scaled values.
    *reinterpret_cast<__half2*>(&params.dst[offset]) = __float22half2_rn(f2);
  }
}

#ifdef PADDLE_CUDA_BF16
template <>
inline __device__ void GroupNormCompute<phi::dtype::bfloat16, 2>(
    int32_t dhwBegin,
    int32_t dhwEnd,
    int32_t ci,
    const GroupNormNDHWCParams<phi::dtype::bfloat16>& params,
    float mean,
    float invStdDev) {
  float2 gammaF2, betaF2;
  gammaF2 = phi::bfloat1622float2(*reinterpret_cast<__hip_bfloat162 const*>(
      reinterpret_cast<__hip_bfloat16 const*>(params.gamma) + ci));
  betaF2 = phi::bfloat1622float2(*reinterpret_cast<__hip_bfloat162 const*>(
      reinterpret_cast<__hip_bfloat16 const*>(params.beta) + ci));

  // Iterate over the activations to compute the sums.
  for (int32_t dhwi = dhwBegin; dhwi < dhwEnd; ++dhwi) {
    // The src/dst offset.
    int64_t offset = (int64_t)blockIdx.z * params.dhwc + dhwi * params.c + ci;

    // Fetch two channels per thread.
    __hip_bfloat162 h2 =
        *reinterpret_cast<__hip_bfloat162 const*>(&params.srcX[offset]);

    // Extract the two half values.
    float2 f2 = phi::bfloat1622float2(h2);
    // 获取残差，如果有的话
    if (params.srcY != nullptr) {
      auto gi = ci / params.cPerGroup;
      auto gj = ci % params.cPerGroup;
      __hip_bfloat162 r2 = *reinterpret_cast<__hip_bfloat162 const*>(
          &params.srcY[gi * params.cPerGroup + gj]);
      float2 r_f2 = phi::bfloat1622float2(r2);
      f2.x += r_f2.x;
      f2.y += r_f2.y;
      *reinterpret_cast<__hip_bfloat162*>(&params.eleOut[offset]) =
          phi::float22bfloat162_rn(f2);
      ;
    }
    // Normalize the channels.
    f2.x = (f2.x - mean) * invStdDev;
    f2.y = (f2.y - mean) * invStdDev;

    // Scale by gamma and add beta.
    f2.x = gammaF2.x * f2.x + betaF2.x;
    f2.y = gammaF2.y * f2.y + betaF2.y;

    // Apply Silu if needed.
    if (params.withSilu) {
      f2.x = f2.x * sigmoid(f2.x);
      f2.y = f2.y * sigmoid(f2.y);
    }
    // Store the scaled values.
    *reinterpret_cast<__hip_bfloat162*>(&params.dst[offset]) =
        phi::float22bfloat162_rn(f2);
  }
}
#endif

template <typename T, int THREADS_PER_CHANNEL>
__global__ void groupNormNDHWCScaleKernel(
    const GroupNormNDHWCParams<T> params) {
  // The instance in the batch.
  int32_t ni = blockIdx.z;
  // The channel loaded by that thread (2 channels per thread for F16x2).
  int32_t ci =
      blockIdx.x * params.cPerBlock + threadIdx.x * THREADS_PER_CHANNEL;

  // The group that thread works on and the channel in the group (modulus).
  int32_t gi = ci / params.cPerGroup;
  // groupsize的 id
  int32_t gj = ci % params.cPerGroup;

  if (ci >= params.c || gi >= params.groups) {
    return;
  }

  // Load the sum and sum of squares for the group.

  float mean = params.redBuffer[(2 * ni + 0) * params.groups + gi];
  float sumSq = params.redBuffer[(2 * ni + 1) * params.groups + gi];

  // Compute the variance.
  float var = sumSq * params.invDHWC - (mean * mean);

  if (params.var_data != nullptr) {
    params.var_data[ni * params.groups + gi] = var;
  }
  // Compute the inverse of the stddev.
  float invStdDev = rsqrtf(var + params.eps);

  // The first activation loaded by that block.
  int32_t dhwBegin = blockIdx.y * params.dhwPerBlock;
  // The last activation loaded by that block.
  int32_t dhwEnd = min(dhwBegin + params.dhwPerBlock, params.dhw);
  GroupNormCompute<T, THREADS_PER_CHANNEL>(
      dhwBegin, dhwEnd, ci, params, mean, invStdDev);
}

template <typename T>
void groupNormNDHWCScale<T>::operator()(const GroupNormNDHWCParams<T>& params,
                                        gpuStream_t stream) {
  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = divUp(params.c, params.cPerBlock);
  // The number of blocks to compute all the activations in a given instance.
  grid.y = divUp(params.dhw, params.dhwPerBlock);
  // The number of instances.
  grid.z = params.n;
  // 如果groupsize能被整除
  if (params.cPerGroup % 2 == 0) {
    switch (params.cPerBlock) {
      case 512:
      case 480:
        groupNormNDHWCScaleKernel<T, 2><<<grid, 256, 0, stream>>>(params);
        break;
      case 320:
        groupNormNDHWCScaleKernel<T, 2><<<grid, 160, 0, stream>>>(params);
        break;
      case 256:
        groupNormNDHWCScaleKernel<T, 2><<<grid, 128, 0, stream>>>(params);
        break;
      case 128:
        groupNormNDHWCScaleKernel<T, 2><<<grid, 64, 0, stream>>>(params);
        break;
      default:
        grid.x = divUp(params.c, 128);
        groupNormNDHWCScaleKernel<T, 2><<<grid, 64, 0, stream>>>(params);
    }
  } else {
    switch (params.cPerBlock) {
      case 512:
        groupNormNDHWCScaleKernel<T, 1><<<grid, 512, 0, stream>>>(params);
        break;
      case 480:
        groupNormNDHWCScaleKernel<T, 1><<<grid, 480, 0, stream>>>(params);
        break;
      case 320:
        groupNormNDHWCScaleKernel<T, 1><<<grid, 320, 0, stream>>>(params);
        break;
      case 256:
        groupNormNDHWCScaleKernel<T, 1><<<grid, 256, 0, stream>>>(params);
        break;
      case 128:
        groupNormNDHWCScaleKernel<T, 1><<<grid, 128, 0, stream>>>(params);
        break;
      default:
        grid.x = divUp(params.c, 128);
        groupNormNDHWCScaleKernel<T, 1><<<grid, 128, 0, stream>>>(params);
    }
  }
}
template class groupNormNDHWCScale<half>;

template <typename T, typename Context>
void GroupNormNDHWCKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const paddle::optional<DenseTensor>& residual,
                          const paddle::optional<DenseTensor>& scale,
                          const paddle::optional<DenseTensor>& bias,
                          float epsilon,
                          int groups,
                          const std::string& data_layout_str,
                          const std::string& activation,
                          DenseTensor* y,
                          DenseTensor* residual_out,
                          DenseTensor* mean,
                          DenseTensor* var) {
  using AccT = typename phi::dtype::MPTypeTrait<T>::Type;
  GroupNormNDHWCParams<T> params_;
  params_.withSilu = activation == "silu" ? true : false;

  const auto x_dims = x.dims();
  dev_ctx.template Alloc<T>(y);
  const T* x_data = x.data<T>();
  const T* residual_data = nullptr;
  const auto residual_ptr = residual.get_ptr();
  T* residual_out_data = nullptr;
  if (residual_ptr) {
    dev_ctx.template Alloc<T>(residual_out);
    residual_data = residual_ptr->data<T>();
    residual_out_data = residual_out->data<T>();
  }
  T* y_data = y->data<T>();
  const auto scale_ptr = scale.get_ptr();
  const auto bias_ptr = bias.get_ptr();
  const T* scale_data = nullptr;
  if (scale_ptr) scale_data = scale_ptr->data<T>();
  const T* bias_data = nullptr;
  if (bias_ptr) bias_data = bias_ptr->data<T>();
  const auto d_dim = x_dims.size();
  params_.n = x_dims[0];
  if (d_dim == 3) {
    params_.c = x_dims[2];
    params_.d = 1;
    params_.h = 1;
    params_.w = x_dims[1];
  } else if (d_dim == 4) {
    params_.c = x_dims[3];
    params_.d = 1;
    params_.h = x_dims[1];
    params_.w = x_dims[2];
  } else {
    // d_dim == 5
    params_.c = x_dims[4];
    params_.d = x_dims[1];
    params_.h = x_dims[2];
    params_.w = x_dims[3];
  }

  dev_ctx.template Alloc<AccT>(mean);
  dev_ctx.template Alloc<AccT>(var);
  auto* mean_data = mean->data<AccT>();
  auto* var_data = var->data<AccT>();
  params_.var_data = var_data;
  // cPerBlock 每一个block中有处理多少个 groupsize
  int32_t cPerBlock = 320;
  int32_t maxBlocksPerDHW = 1024;
  switch (params_.c) {
    case 2048:
    case 1024:
      cPerBlock = 512;
      break;
    case 960:
    case 1920:
      cPerBlock = 480;
      break;
    case 512:
    case 256:
      cPerBlock = 256;
      break;
    case 128:
      cPerBlock = 128;
      break;
    default:
      cPerBlock = 320;
  }
  params_.groups = groups;
  // 每个c有多少个group_size
  // 如果c 不能被group整除呢？
  params_.cPerGroup = params_.c / params_.groups;
  // 如果cPerBlock 能被 group_size 整除就直接使用 ，否则 换成 group_size
  // cPerBlock 每一个block中有处理多少个 groupsize
  // 需要用多少个block 处理 group size  感觉是把c按什么大小分割
  if (cPerBlock % params_.cPerGroup != 0) {
    cPerBlock = params_.cPerGroup;
  }
  params_.srcX = reinterpret_cast<const T*>(x_data);
  params_.dst = reinterpret_cast<T*>(y_data);
  // 增加 残差
  if (residual_ptr) {
    params_.srcY = reinterpret_cast<const T*>(residual_data);
    params_.eleOut = reinterpret_cast<T*>(residual_out_data);
  }
  params_.gamma = scale_data;
  params_.beta = bias_data;
  params_.dhw = params_.d * params_.h * params_.w;
  // 不大于 1024的 dhw 能整除的数，最大是1024,  blocksPerDHW
  // 感觉是dhw需要用多少个block处理,但是这个blockperdhw 是dhw的除数呀
  const int32_t blocksPerDHW = findMaxDivisor(params_.dhw, maxBlocksPerDHW);
  // 每个block中有需要处理多少个dhw
  params_.dhwPerBlock = divUp(params_.dhw, blocksPerDHW);
  params_.cPerBlock = cPerBlock;
  params_.dhwc = params_.dhw * params_.c;
  // 所以是每个block需要处理的是 dhw*group_size个数据
  params_.invDHWC = 1.F / static_cast<float>(params_.dhw * params_.cPerGroup);
  params_.eps = epsilon;
  auto stream = dev_ctx.stream();
  DenseTensor redBuffer;
  int buffer_sizes = 2 * params_.n * groups;
  redBuffer.Resize({1, buffer_sizes});
  params_.redBuffer = dev_ctx.template Alloc<float>(&redBuffer);
#ifdef PADDLE_WITH_HIP
  hipMemset(params_.redBuffer, 0, buffer_sizes * sizeof(float));
#else
  hipMemset(params_.redBuffer, 0, buffer_sizes * sizeof(float));
#endif
  groupNormNDHWCSum<T> ndhwc_sum;
  ndhwc_sum(&params_, stream);
  groupNormNDHWCScale<T> ndhwc_scale;
  ndhwc_scale(params_, stream);
#ifdef PADDLE_WITH_HIP
  phi::backends::gpu::GpuMemcpyAsync(mean_data,
                                     params_.redBuffer,
                                     params_.n * groups * sizeof(float),
                                     hipMemcpyDeviceToHost,
                                     stream);
#else
  phi::backends::gpu::GpuMemcpyAsync(mean_data,
                                     params_.redBuffer,
                                     params_.n * groups * sizeof(float),
                                     hipMemcpyDeviceToHost,
                                     stream);
#endif
}

// grid(group_size,groups,n)
// threads(min(hw,1024),1,1)
template <typename T, typename AccT>
__global__ void GroupNormForwardGetMeanAndVar(const T* x,
                                              const T* residual_data,
                                              int N,
                                              int C,
                                              int W,
                                              int imsize,  // h*w
                                              int groups,
                                              int group_size,
                                              AccT* mean,
                                              AccT* var) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  // 如果c没有办法被 group除尽， 那么C - gid * group_size 会有遗留时 number是
  // 每次处理的是groupsize 或者是 remain
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  // c部分 正在处理的 block是 第几个g 的第几个 g_size
  int ccid = gid * group_size + cid;
  // 防止大于c,针对遗留的部分
  if (ccid >= C) return;
  AccT x_mean = static_cast<AccT>(0);
  AccT x_var = static_cast<AccT>(0);
  // threadId min(hw,1024) imsize=h*w？ 为什么h和w不固定 为什么线程不固定
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    AccT val;
    int hid = imid / W;
    int wid = imid % W;
    val = static_cast<AccT>(x[(bid * H + hid) * W * C + wid * C + ccid]);
    if (residual_data != nullptr) {
      val += static_cast<AccT>(residual_data[ccid]);
    }
    x_mean += val;
    x_var += val * val;
  }
  x_mean /= number * imsize;
  x_var /= number * imsize;
  CudaAtomicAddWithWarp(&mean[bid * groups + gid], x_mean);
  CudaAtomicAddWithWarp(&var[bid * groups + gid], x_var);
}

// grid [group_size,groups,n]
// thread [min(hw,1024,1,1)]
template <typename T, typename AccT, int flags>
__global__ void GroupNormForward(const T* x,
                                 const T* residual,
                                 const AccT* mean,
                                 const AccT* var,
                                 const T* scale,
                                 const T* bias,
                                 int N,
                                 int C,
                                 int W,
                                 int imsize,
                                 int groups,
                                 int group_size,
                                 AccT epsilon,
                                 T* y,
                                 T* residual_out,
                                 AccT* real_var,
                                 const DataLayout data_layout,
                                 bool with_silu) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  auto ng = bid * groups + gid;
  AccT x_mean = mean[ng];
  AccT x_var = var[ng];
  x_var = x_var - x_mean * x_mean;

  AccT var_inv = rsqrt(x_var + epsilon);
  if (cid == 0 && threadIdx.x == 0) {
    real_var[ng] = x_var;
  }
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    AccT val;
    int hid, wid;
    int index = (bid * C + ccid) * imsize + imid;
    if (data_layout == DataLayout::kNCHW) {
      val = static_cast<AccT>(x[index]);
    } else {
      hid = imid / W;
      wid = imid % W;
      val = static_cast<AccT>(x[(bid * H + hid) * W * C + wid * C + ccid]);
    }
    if (residual != nullptr) {
      val += static_cast<AccT>(residual[ccid]);
    }
    if (data_layout == DataLayout::kNCHW) {
      residual_out[index] = static_cast<T>(val);
    } else {
      residual_out[(bid * H + hid) * W * C + wid * C + ccid] =
          static_cast<T>(val);
    }
    val = (val - x_mean) * var_inv;
    if (flags & kHasScale) {
      val *= static_cast<AccT>(scale[ccid]);
    }
    if (flags & kHasBias) {
      val += static_cast<AccT>(bias[ccid]);
    }
    // Apply Silu if needed.
    if (with_silu) {
      val = val * sigmoid(val);
    }
    if (data_layout == DataLayout::kNCHW) {
      y[index] = static_cast<T>(val);
    } else {
      y[(bid * H + hid) * W * C + wid * C + ccid] = static_cast<T>(val);
    }
  }
}

template <typename T, typename AccT>
void GroupNormDirectCUDAFunctor<T, AccT>::operator()(
    gpuStream_t stream,
    const T* input,
    std::vector<int> input_shape,
    const T* bias,
    const T* scale,
    AccT* temp_variance,
    int groups,
    float eps,
    T* output,
    AccT* mean,
    AccT* variance,
    const DataLayout data_layout) {
  const auto input_ddim = common::make_ddim(input_shape);
  const int C =
      (data_layout == DataLayout::kNCHW ? input_ddim[1]
                                        : input_ddim[input_ddim.size() - 1]);
  const int group_size = C / groups;
  const int W =
      (data_layout == DataLayout::kNCHW ? input_ddim[input_ddim.size() - 1]
                                        : input_ddim[input_ddim.size() - 2]);

  int image_size = 1;
  if (data_layout == DataLayout::kNCHW) {
    for (int i = 2; i < input_ddim.size(); ++i) {
      image_size *= input_ddim[i];
    }
  } else {
    for (int i = 1; i < input_ddim.size() - 1; ++i) {
      image_size *= input_ddim[i];
    }
  }
  int block_size = std::min(1024, image_size);
  dim3 grid(group_size, groups, input_ddim[0]);
  dim3 threads(block_size, 1, 1);
  if (data_layout == DataLayout::kNCHW) {
    constexpr int vec_size = sizeof(float4) / sizeof(T);
    int size = group_size * image_size;  // group element size
    const int max_num_threads = 1024;
    int max_block_size = std::min(size / vec_size, max_num_threads);
    int block_size_nchw = 1;
    while (block_size_nchw < max_block_size) {
      block_size_nchw *= 2;
    }

    block_size_nchw = std::max(block_size_nchw, phi::kps::details::kWarpSize);
    dim3 grids(input_ddim[0] * groups);
    dim3 blocks(block_size_nchw);

    if (size < vec_size * block_size_nchw) {
      phi::ScalarGetMeanAndVarNCHW<T, AccT><<<grids, blocks, 0, stream>>>(
          input,nullptr, mean, temp_variance, size, input_ddim[0], group_size);
    } else {
      phi::VectorizedGetMeanAndVarNCHW<T, AccT, vec_size>
          <<<grids, blocks, 0, stream>>>(
              input,nullptr, mean, temp_variance, size, input_ddim[0], group_size);
    }
  } else {
#ifdef PADDLE_WITH_HIP
    hipMemset(mean, 0, sizeof(AccT) * input_ddim[0] * groups);
    hipMemset(temp_variance, 0, sizeof(AccT) * input_ddim[0] * groups);
#else
    hipMemset(mean, 0, sizeof(AccT) * input_ddim[0] * groups);
    hipMemset(temp_variance, 0, sizeof(AccT) * input_ddim[0] * groups);
#endif

    phi::GroupNormForwardGetMeanAndVar<T, AccT>
        <<<grid, threads, 0, stream>>>(input,
                                       nullptr,
                                       input_ddim[0],
                                       C,
                                       W,
                                       image_size,
                                       groups,
                                       group_size,
                                       mean,
                                       temp_variance);
  }
  GroupNormForward<T, AccT, 3>
      <<<grid, threads, 0, stream>>>(input,
                                     nullptr,
                                     mean,
                                     temp_variance,
                                     scale,
                                     bias,
                                     input_ddim[0],
                                     C,
                                     W,
                                     image_size,
                                     groups,
                                     group_size,
                                     static_cast<AccT>(eps),
                                     output,
                                     nullptr,
                                     variance,
                                     data_layout,
                                     false);
}
template class GroupNormDirectCUDAFunctor<float, float>;
#if defined(PADDLE_WITH_CUDA) && !defined(PADDLE_WITH_HIP)
template class GroupNormDirectCUDAFunctor<half, float>;
#endif

template <typename T, typename Context>
void GroupNormGeneralCaseKernel(const Context& dev_ctx,
                                const DenseTensor& x,
                                const paddle::optional<DenseTensor>& residual,
                                const paddle::optional<DenseTensor>& scale,
                                const paddle::optional<DenseTensor>& bias,
                                float epsilon,
                                int groups,
                                const std::string& data_layout_str,
                                const std::string& activation,
                                DenseTensor* y,
                                DenseTensor* residual_out,
                                DenseTensor* mean,
                                DenseTensor* var) {
  using AccT = typename phi::dtype::MPTypeTrait<T>::Type;
  const auto with_silu = activation == "silu" ? true : false;
  const DataLayout data_layout = common::StringToDataLayout(data_layout_str);
  const auto residual_ptr = residual.get_ptr();
  const auto scale_ptr = scale.get_ptr();
  const auto bias_ptr = bias.get_ptr();
  const auto x_dims = x.dims();
  const int C = (data_layout == DataLayout::kNCHW ? x_dims[1]
                                                  : x_dims[x_dims.size() - 1]);
  const int group_size = C / groups;
  const int W = (data_layout == DataLayout::kNCHW ? x_dims[x_dims.size() - 1]
                                                  : x_dims[x_dims.size() - 2]);

  dev_ctx.template Alloc<T>(y);
  dev_ctx.template Alloc<AccT>(mean);
  dev_ctx.template Alloc<AccT>(var);
  // temp_var is used to calculate the mean^2
  DenseTensor temp_var;
  temp_var.Resize(var->dims());
  dev_ctx.template Alloc<AccT>(&temp_var);
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  phi::funcs::SetConstant<GPUContext, AccT> set_zero_AccT;
  auto* x_data = x.data<T>();
  auto* y_data = y->data<T>();
  auto* mean_data = mean->data<AccT>();
  auto* var_data = var->data<AccT>();
  auto* temp_var_data = temp_var.data<AccT>();

  const T* scale_data = nullptr;
  if (scale_ptr) scale_data = scale_ptr->data<T>();
  const T* bias_data = nullptr;
  if (bias_ptr) bias_data = bias_ptr->data<T>();
  const T* residual_data = nullptr;
  T* residual_out_data = nullptr;
  if (residual_ptr) {
    residual_data = residual_ptr->data<T>();
    dev_ctx.template Alloc<T>(residual_out);
    residual_out_data = residual_out->data<T>();
  }
  int imsize = 1;
  if (data_layout == DataLayout::kNCHW) {
    for (int i = 2; i < x_dims.size(); ++i) {
      imsize *= x_dims[i];
    }
  } else {
    for (int i = 1; i < x_dims.size() - 1; ++i) {
      imsize *= x_dims[i];
    }
  }

  int block_size = std::min(1024, imsize);

  dim3 grid(group_size, groups, x_dims[0]);
  dim3 threads(block_size, 1, 1);
  if (data_layout == DataLayout::kNCHW) {
    constexpr int vec_size = sizeof(float4) / sizeof(T);
    int size = group_size * imsize;
    const int max_num_threads = 1024;
    int max_block_size = std::min(size / vec_size, max_num_threads);
    int block_size_nchw = 1;
    while (block_size_nchw < max_block_size) {
      block_size_nchw *= 2;
    }
    block_size_nchw = std::max(block_size_nchw, kps::details::kWarpSize);
    dim3 grids(x_dims[0] * groups);
    dim3 blocks(block_size_nchw);
    if (size < vec_size * block_size_nchw) {
      ScalarGetMeanAndVarNCHW<T, AccT>
          <<<grids, blocks, 0, dev_ctx.stream()>>>(x_data,
                                                   residual_data,
                                                   mean_data,
                                                   temp_var_data,
                                                   size,
                                                   x_dims[0],
                                                   group_size);
    } else {
      VectorizedGetMeanAndVarNCHW<T, AccT, vec_size>
          <<<grids, blocks, 0, dev_ctx.stream()>>>(x_data,
                                                   residual_data,
                                                   mean_data,
                                                   temp_var_data,
                                                   size,
                                                   x_dims[0],
                                                   group_size);
    }
  } else {
    set_zero_AccT(dev_ctx, mean, static_cast<AccT>(0));
    set_zero_AccT(dev_ctx, &temp_var, static_cast<AccT>(0));
    GroupNormForwardGetMeanAndVar<T, AccT>
        <<<grid, threads, 0, dev_ctx.stream()>>>(x_data,
                                                 residual_data,
                                                 x_dims[0],
                                                 C,
                                                 W,
                                                 imsize,
                                                 groups,
                                                 group_size,
                                                 mean_data,
                                                 temp_var_data);
  }
  int flags =
      (scale_data != nullptr) * kHasScale + (bias_data != nullptr) * kHasBias;
  UNROLL_ALL_CASES(flags,
                   GroupNormForward,
                   x_data,
                   residual_data,
                   mean_data,
                   temp_var_data,
                   scale_data,
                   bias_data,
                   x_dims[0],
                   C,
                   W,
                   imsize,
                   groups,
                   group_size,
                   static_cast<AccT>(epsilon),
                   y_data,
                   residual_out_data,
                   var_data,
                   data_layout,
                   with_silu);
}

template <typename T, typename Context>
void GroupNormKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const paddle::optional<DenseTensor>& scale,
                     const paddle::optional<DenseTensor>& bias,
                     float epsilon,
                     int groups,
                     const std::string& data_layout_str,
                     DenseTensor* y,
                     DenseTensor* mean,
                     DenseTensor* var) {
  using std::is_same;
  if (is_same<T, phi::dtype::float16>::value && data_layout_str == "NHWC") {
    GroupNormNDHWCKernel<phi::dtype::float16, Context>(dev_ctx,
                                                       x,
                                                       nullptr,
                                                       scale,
                                                       bias,
                                                       epsilon,
                                                       groups,
                                                       data_layout_str,
                                                       "",
                                                       y,
                                                       nullptr,
                                                       mean,
                                                       var);
    return;
  }

#ifdef PADDLE_CUDA_BF16
  if (is_same<T, phi::dtype::bfloat16>::value && data_layout_str == "NHWC") {
    GroupNormNDHWCKernel<phi::dtype::bfloat16, Context>(dev_ctx,
                                                        x,
                                                        nullptr,
                                                        scale,
                                                        bias,
                                                        epsilon,
                                                        groups,
                                                        data_layout_str,
                                                        "",
                                                        y,
                                                        nullptr,
                                                        mean,
                                                        var);
    return;
  }
#endif

  GroupNormGeneralCaseKernel<T, Context>(dev_ctx,
                                         x,
                                         residual,
                                         scale,
                                         bias,
                                         epsilon,
                                         groups,
                                         data_layout_str,
                                         activation,
                                         y,
                                         residual_out,
                                         mean,
                                         var);
}

}  // namespace phi

PD_REGISTER_KERNEL(group_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::GroupNormKernel,
                   float,
                   double,
                   phi::dtype::bfloat16,
                   phi::dtype::float16) {
  if (kernel_key.dtype() == phi::DataType::BFLOAT16 ||
      kernel_key.dtype() == phi::DataType::FLOAT16) {
    kernel->OutputAt(1).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(2).SetDataType(phi::DataType::FLOAT32);
  }
}

PD_REGISTER_KERNEL(add_group_norm_silu,
                   GPU,
                   ALL_LAYOUT,
                   phi::GroupNormNDHWCKernel,
                   phi::dtype::bfloat16,
                   phi::dtype::float16) {
    kernel->OutputAt(1).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(2).SetDataType(phi::DataType::FLOAT32);
}
